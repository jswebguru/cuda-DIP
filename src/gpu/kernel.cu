#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <valarray>
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/gpu/gpu.hpp"

__global__ void foo(int *a, int N) {
 int i=blockIdx.x*blockDim.x+threadIdx.x;
    a[i]=i;
}

int test_kernel()
{
  int N=4097;
  int threads=128;
  int blocks=(N+threads-1)/threads;
  int *a;

  hipMallocManaged(&a,N * sizeof (int));
  foo<<<blocks,threads>>>(a, N);
  hipDeviceSynchronize();

  for (int i=0;i<10;i++)
    printf("%d\n",a[i]);

  return 0;
}

int main(int argc, char** argv)
{
    test_kernel();
    return 0;
}

