#include <stdio.h>
#include <iostream>
#include <valarray>
#include <assert.h>
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "kernel.cuh"
#define TILE_WIDTH_PIX 4
#define TILE_WIDTH 16
#define TILE_HEIGHT 16
#define STREL_SIZE 5
#define R (STREL_SIZE / 2)
#define BLOCK_W (TILE_WIDTH + (2 * R))
#define BLOCK_H (TILE_HEIGHT + (2 * R))

int main(int argc, char** argv)
{
    if (argc < 4)
    {
        std::cout << "usage: main <Image_Path> <Func_name> <Conv_size>" << std::endl;
        return 1;
    }
    cv::Mat image;
    image = cv::imread(argv[1], CV_LOAD_IMAGE_UNCHANGED);
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return 1;
    }

    std::string func_name = argv[2];
    int width = image.rows;
    int height = image.cols;

    Rgb* device_dst = empty_img_device(image);
    Rgb* device_img = img_to_device(image);
    Rgb* out = (Rgb*)malloc(width * height * sizeof (Rgb));

    dim3 blockSize;
    if (func_name == "pixelize")
        blockSize = dim3(TILE_WIDTH_PIX, TILE_WIDTH_PIX);
    else
        blockSize = dim3(TILE_WIDTH, TILE_WIDTH);
    int bx = (width + blockSize.x - 1) / blockSize.x;
    int by = (height + blockSize.y - 1) / blockSize.y;
    dim3 gridSize = dim3(bx, by);

    if (func_name == "pixelize")
        kernel_pixelize<<<gridSize, blockSize>>>(device_dst, device_img, width, height, std::stoi(argv[3]));
    else if (func_name == "conv")
        kernel_conv<<<gridSize, blockSize>>>(device_dst, device_img, width, height, std::stoi(argv[3]));
    else if (func_name == "shared_conv")
    {
        dim3 block(16 + STREL_SIZE - 1, 16 + STREL_SIZE - 1);
        dim3 grid(width / (block.x) + block.x, height / (block.y) + block.y);
        kernel_shared_conv<<<grid, block>>>(device_dst, device_img, width, height);
    }
    else
    {
        std::cout << "error: function name '" << func_name << "' is not known." << std::endl;
        hipFree(device_dst);
        hipFree(device_img);
        free(out);
        return 1;
    }

    hipDeviceSynchronize();
    hipMemcpy(out, device_dst, height * width * sizeof (Rgb), hipMemcpyDeviceToHost);

    device_to_img(out, image);

    hipFree(device_dst);
    hipFree(device_img);
    free(out);

    cv::namedWindow("Display Window", CV_WINDOW_AUTOSIZE);
    cv::imshow("Display Window", image);
    cv::waitKey(0);
    return 0;
}
